#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>
#include <Windows.h>
#include "hip/hip_runtime.h"
#include ""
#include <cstdlib>
#include <stdio.h>
#include <iostream>
#include <math.h>
#include <vector>
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>
#include <thrust/device_vector.h>
#include "gputimer.h"
#include <thrust/transform.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/replace.h>
#include <thrust/functional.h>
#include <iostream>
#include <thrust/sort.h>
#include <hiprand.h>
#include <math.h>
#include <hiprand.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <assert.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <map>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}
struct BoundaryConditions
{
	float x0 = 1;
	float y0 = 1;
	float psi0 = 1;
	float cte0 = 1;
	float e0 = 1;
	float l = 1;
	float a0 = 1;
	float a1 = 1;
	float a2 = 1;
	float dt = 0.1;
};
struct Weights
{
	float wCTE = 1;
	float wE = 1;
	float wO = 1;
	float wV = 1;
	float wDo = 1;
	float wDv = 1;
	float wC = 1;
	float Vmax = 1;
};
__global__ void fitness(float* commands, float* fitness, int nSteps, BoundaryConditions boundaryConditions,
	Weights weights)
{
	int k = threadIdx.x + blockDim.x * blockIdx.x;

	//Initial conditions
	float x = boundaryConditions.x0;
	float y = boundaryConditions.y0;
	float psi = boundaryConditions.psi0;
	float cte = boundaryConditions.cte0;
	float e = boundaryConditions.e0;
	float a2 = boundaryConditions.a2;
	float a1 = boundaryConditions.a1;
	float a0 = boundaryConditions.a0;
	float dt = boundaryConditions.dt;
	float l = boundaryConditions.l;

	float cost = 0;

	//derrivatives
	float xD = 0;
	float yD = 0;
	float psiD = 0;
	float lastAngle = 0;
	float lastSpeed = 0;

	for (int i = 0; i < nSteps; i++)
	{
		int specimenOffset = nSteps * 2 * k;
		float speed = commands[specimenOffset + 2 * i];
		float angle = commands[specimenOffset + 2 * i + 1];

		//error
		cte = a2 * x * x + a1 * x + a0 - y;
		e = atan(2 * a2 * x + a1);

		//state update
		xD = speed * cos(psi);
		yD = speed * sin(psi);
		psiD = speed * angle / l;
		x += xD * dt;
		y += yD * dt;
		psi += psiD * dt;

		cost +=
			weights.wCTE * pow(cte, 2) +
			weights.wE * pow(e, 2) +
			weights.wO * pow(angle, 2) +
			weights.wV * pow(weights.Vmax - speed, 2) +
			weights.wDo * pow(angle - lastAngle, 2) +
			weights.wDv * pow(speed - lastSpeed, 2) +
			weights.wC * pow(speed * angle, 2);

		lastAngle = angle;
		lastSpeed = speed;
	}
	fitness[k] = cost;

}

__global__ void worstHalf(float* fitness, int* indexes)
{

}
class Population
{
public:
	Population(int populationSize, int numberOfSteps, int numberOfGenerations, float chanceOfMutation, float chanceOfCrossover)
	{
		this->populationSize = populationSize;
		this->numberOfSteps = numberOfSteps;
		this->numberOfGenerations = numberOfGenerations;
		this->chanceOfMutation = chanceOfMutation;
		this->chanceOfCrossover = chanceOfCrossover;
	}
	Population()
	{
		this->populationSize = 1000;
		this->numberOfSteps = 10;
		this->numberOfGenerations = 100;
		this->chanceOfMutation = 0.05;
		this->chanceOfCrossover = 1;
	}
	void initializePopulation()
	{
		srand(time(NULL));
		gpuErrchk(hipMalloc((void**)& d_commands, 2 * 10 * populationSize * sizeof(float)));//commands
		float* h_commands = new float[2 * 10 * populationSize];
		for (int i = 0; i < 20 * populationSize; i++)
		{
			h_commands[i] = -0.44 + (float)(rand() % 88) / 100;
		}
		gpuErrchk(hipMalloc((void**)& d_fitness, populationSize * sizeof(float)));
		hipMemcpy(d_commands, h_commands, 20 * populationSize * sizeof(float), hipMemcpyHostToDevice);

		fitness << <10, 100 >> > (d_commands, d_fitness, 10, boundaryConditions, weights);

		float* h_fitness = new float[populationSize];
		h_h = new float[populationSize];
		gpuErrchk(hipMemcpy(h_fitness, d_fitness, populationSize * sizeof(float), hipMemcpyDeviceToHost));
		for (int i = 0; i < populationSize * 20; i++)
		{
			//std::cout << h_fitness[i] << std::endl;
		}
		worstIndexes(h_fitness);
	}
	int* worstIndexes(float* fitness)
	{
		const int N = 6;
		int    keys[N] = { 1,   4,   2,   8,   5,   7 };
		char values[N] = { 'a', 'b', 'c', 'd', 'e', 'f' };
		thrust::sort_by_key(keys, keys + N, values);
		// keys is now   {  1,   2,   4,   5,   7,   8}
		// values is now {'a', 'c', 'b', 'e', 'f', 'd'}
		int* vals = new int[populationSize];
		int*  fit = new int[populationSize];
		for (int i = 0; i < populationSize; i++)
		{
			vals[i] = i;
			fit[i] = rand() % 20;
		}
		//for (int i = 0; i < populationSize; i++)
		//	printf("%f %d \n", fitness[i], vals[i]);// = i;
		//printf("--------------\n");
		//thrust::sort_by_key(fitness, fitness + populationSize, vals);
		//for (int i = 0; i < populationSize; i++)
		//	printf("%f %d \n",fitness[i], vals[i]);// = i;
		return NULL;
	}
	float* d_d;
	float* h_h;
	int populationSize = 1000;
	int numberOfSteps = 10;
	int numberOfGenerations = 100;
	float chanceOfMutation = 0.05;
	float chanceOfCrossover = 1;
	float* d_commands;
	float* d_fitness;
	BoundaryConditions boundaryConditions;
	Weights weights;
private:

};


int main()
{
	GpuTimer timer;
	timer.Start();
	Population ne;
	ne.initializePopulation();
	timer.Stop();
	printf("%f",timer.Elapsed());

	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());

	return 0;
}
