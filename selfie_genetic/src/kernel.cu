#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>
#include <fstream>
#include <string>
#include ""
#include <cstdlib>
#include <stdio.h>
#include <iostream>
#include <math.h>
#include <vector>
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>
#include <thrust/device_vector.h>
#include "gputimer.h"
#include <thrust/transform.h>
#include <thrust/execution_policy.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/replace.h>
#include <thrust/functional.h>
#include <iostream>
#include <thrust/sort.h>
#include <hiprand.h>
#include <math.h>
#include <hiprand.h>
#include <hip/device_functions.h>
#include "hip/hip_runtime_api.h"
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <assert.h>
#include <string>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <ros/ros.h>
#include <std_msgs/Float64.h>
#include <std_msgs/Float32.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}
struct BoundaryConditions
{
	float x0 = 1;
	float y0 = 1;
	float psi0 = 1;
	float cte0 = 1;
	float e0 = 1;
	float l = 1;
	float a0 = 1;
	float a1 = 1;
	float a2 = 1;
	float dt = 0.1;
};
struct Weights
{
	float wCTE = 1;
	float wE = 1;
	float wO = 1;
	float wV = 1;
	float wDo = 1;
	float wDv = 1;
	float wC = 1;
	float Vmax = 1;
};
__global__ void fitness(float* commands, float* fitness, int nSteps, BoundaryConditions boundaryConditions, Weights weights)
{
	int k = threadIdx.x + blockDim.x * blockIdx.x;

	//Initial conditions
	float x = boundaryConditions.x0;
	float y = boundaryConditions.y0;
	float psi = boundaryConditions.psi0;
	float cte = boundaryConditions.cte0;
	float e = boundaryConditions.e0;
	float a2 = boundaryConditions.a2;
	float a1 = boundaryConditions.a1;
	float a0 = boundaryConditions.a0;
	float dt = boundaryConditions.dt;
	float l = boundaryConditions.l;

	float cost = 0;

	//derrivatives
	float xD = 0;
	float yD = 0;
	float psiD = 0;
	float lastAngle = 0;
	float lastSpeed = 0;

	for (int i = 0; i < nSteps; i++)
	{
		int specimenOffset = nSteps * 2 * k;
		float speed = commands[specimenOffset + 2 * i];
		float angle = commands[specimenOffset + 2 * i + 1];

		//error
		cte = a2 * x * x + a1 * x + a0 - y;
		e = atan(2 * a2 * x + a1);

		//state update
		xD = speed * cos(psi);
		yD = speed * sin(psi);
		psiD = speed * angle / l;
		x += xD * dt;
		y += yD * dt;
		psi += psiD * dt;

		cost +=
			weights.wCTE * pow(cte, 2) +
			weights.wE * pow(e, 2) +
			weights.wO * pow(angle, 2) +
			weights.wV * pow(weights.Vmax - speed, 2) +
			weights.wDo * pow(angle - lastAngle, 2) +
			weights.wDv * pow(speed - lastSpeed, 2) +
			weights.wC * pow(speed * angle, 2);

		lastAngle = angle;
		lastSpeed = speed;
	}
	fitness[k] = cost;
#ifdef DEBUGCOST
	printf("Koszt wynosi: %f\n", fitness[k]);
#endif // DEBUG

}
__global__ void sortWithIndexes(float* fitness, int* indexes, int populationSize)//dziala
{
#ifdef DEBUGSORT
	for (int i = 0; i < populationSize; i++)
		printf("Indeks i fitness przed sortowaniem: %d %f\n", indexes[i], fitness[i]);
#endif // DEBUGSORT

	thrust::sort_by_key(thrust::device, fitness, fitness + populationSize, indexes);
#ifdef DEBUGSORT
	for (int i = 0; i < populationSize; i++)
		printf("Indeks i fitness po sortowaniu: %d %f\n", indexes[i], fitness[i]);
#endif // DEBUGSORT
}
__global__ void printCommands(float* commands)
{
	int k = threadIdx.x + blockDim.x * blockIdx.x;
	for (int i = 0; i < 20; i++)
	{
		if ((k * 20 + i) % 2 == 0)
			printf("skret %f\n", commands[k * 20 + i]);
		else
			printf("predkosc %f\n", commands[k * 20 + i]);
	}
}
__global__ void crossover(float* commands, int* indexes, int* random)
{
	int k = threadIdx.x + blockDim.x * blockIdx.x;
	for (int i = 0; i < 10; i++)
	{
		int rand1 = random[k]; __syncthreads();
		int rand2 = random[500 + k]; __syncthreads(); //random[0,500]
		int parentOffset1 = 20 * indexes[rand1]; __syncthreads();//k jest z zakresu 0-50%pop(najlepsza cz�� populacji)
		int parentOffset2 = 20 * indexes[rand2]; __syncthreads();//k jest z zakresu 0-50%pop(najlepsza cz�� populacji)
		if (i < (k % 10))
		{
			float speed = commands[parentOffset1 + 2 * i + 1];
			__syncthreads();
			float angle = commands[parentOffset1 + 2 * i]; __syncthreads();
			//printf("%d \n", 500 + k * 20 + 2 * i + 1);
			commands[(500 + k) * 20 + 2 * i + 1] = speed; __syncthreads();
			commands[(500 + k) * 20 + 2 * i] = angle; __syncthreads();
		}
		else
		{
			float speed = commands[parentOffset2 + 2 * i + 1]; __syncthreads();
			float angle = commands[parentOffset2 + 2 * i]; __syncthreads();
			//printf("%d \n", 500 + k * 20 + 2 * i + 1);
			commands[(500 + k) * 20 + 2 * i + 1] = speed; __syncthreads();
			commands[(500 + k) * 20 + 2 * i] = angle; __syncthreads();
		}
	};
}
int* generateRandom(int min, int max, int size)
{
	srand(time(NULL));
	int* random = new int[size];
	for (int i = 0; i < size; i++)
	{
		random[i] = min + rand() % max;
	}
	return random;
}


class Population
{
public:
	Population(int populationSize, int numberOfSteps, int numberOfGenerations, float chanceOfMutation, float chanceOfCrossover)
	{
		this->populationSize = populationSize;
		this->numberOfSteps = numberOfSteps;
		this->numberOfGenerations = numberOfGenerations;
		this->chanceOfMutation = chanceOfMutation;
		this->chanceOfCrossover = chanceOfCrossover;
	}
	Population()
	{
		this->populationSize = 1000;
		this->numberOfSteps = 10;
		this->numberOfGenerations = 100;
		this->chanceOfMutation = 0.05;
		this->chanceOfCrossover = 1;

	}
	void initializePopulation()
	{
		srand(time(NULL));

		int* vals_temp = new int[populationSize];
		h_commands = new float[2 * 10 * populationSize];
		for (int i = 0; i < populationSize; i++)
		{
			vals_temp[i] = i;
		}
		for (int i = 0; i < 20 * populationSize; i++)
		{
			if (i % 2 == 0)
				h_commands[i] = -0.44 + (float)(rand() % 88) / 100;
			else
				h_commands[i] = -2 + (float)(rand() % 400) / 100;
		}

		gpuErrchk(hipMalloc((void**)& d_vals, populationSize * sizeof(int)));
		gpuErrchk(hipMalloc((void**)& d_commands, 2 * 10 * populationSize * sizeof(float)));//commands
		gpuErrchk(hipMalloc((void**)& d_fitness, populationSize * sizeof(float)));

		gpuErrchk(hipMemcpy(d_vals, vals_temp, populationSize * sizeof(int), hipMemcpyHostToDevice));
		gpuErrchk(hipMemcpy(d_commands, h_commands, 20 * populationSize * sizeof(float), hipMemcpyHostToDevice));
#ifdef DEBUGCOMMANDS
		for (int i = 0; i < 20 * populationSize - 1; i++)
		{
			printf("%f;%f\n", h_commands[i], h_commands[i + 1]);
		}
#endif // DEBUGCOMMANDS
	}
	void dumpLog(std::ofstream& plik)
	{
		for (int j = 0; j < 10; j++)
			plik << "angle;";
		plik << "NULL;";
		for (int j = 0; j < 10; j++)
			plik << "speed;";
		plik << "\n";
		for (int i = 0; i < populationSize; i++)
		{
			plik << "=" << i << ";";
			for (int j = 0; j < 10; j++)
				plik << "=" << h_commands[i * 20 + j * 2] << ";";
			plik << "NULL;";
			for (int j = 0; j < 10; j++)
				plik << "=" << h_commands[i * 20 + j * 2 + 1] << ";";
			plik << "\n";
		}
	}
	void geneticAlgorithm()
	{

		int* randomNumbers = new int[1000];
		randomNumbers = generateRandom(0, 500, 1000);
		int* d_randomNumbers;
		hipMalloc((void**)& d_randomNumbers, 1000 * sizeof(int));
		hipMemcpy(d_randomNumbers, randomNumbers, 1000 * sizeof(int), hipMemcpyHostToDevice);
		fitness << <10, 100 >> > (d_commands, d_fitness, 10, boundaryConditions, weights);
		sortWithIndexes << <1, 1 >> > (d_commands, d_vals, populationSize);


#ifdef DUMPLOG
		gpuErrchk(hipMemcpy(h_commands, d_commands, 10 * 2 * populationSize * sizeof(float), hipMemcpyDeviceToHost));
		std::ofstream plik("dzial.csv");
		dumpLog(plik);
#endif // DUMPLOG
		crossover << <10, 50 >> > (d_commands, d_vals, d_randomNumbers);
#ifdef DUMPLOG
		gpuErrchk(hipMemcpy(h_commands, d_commands, 10 * 2 * populationSize * sizeof(float), hipMemcpyDeviceToHost));
		dumpLog(plik);
#endif // DUMPLOG

	}
	float* d_d;
	float* h_h;
	float* h_commands;
	int populationSize = 1000;
	int numberOfSteps = 10;
	int numberOfGenerations = 100;
	float chanceOfMutation = 0.05;
	float chanceOfCrossover = 1;
	float* d_commands;
	float* d_fitness;
	int* d_vals;
	BoundaryConditions boundaryConditions;
	Weights weights;
private:

};

std_msgs::Float64 steering_angle_msg;
std_msgs::Float64 target_speed_msg;

int main(int argc, char** argv)
{
	ros::init(argc, argv, "genetic_node");
	ros::NodeHandle nh;
	ros::Publisher steering_angle = nh.advertise<std_msgs::Float64>("steering_angle", 1000);
	ros::Publisher target_speed = nh.advertise<std_msgs::Float64>("target_speed", 1000);
	//
	GpuTimer timer;
	Population ne;
	timer.Start();
	ne.initializePopulation();
	timer.Stop();
	printf("%f\n", timer.Elapsed());
	timer.Start();
	for (int i = 0; i < 1; i++)
		ne.geneticAlgorithm();
	timer.Stop();
	printf("%f\n", timer.Elapsed());
	gpuErrchk(hipGetLastError());
	gpuErrchk(hipDeviceSynchronize());
	//
	ros::Rate rate(10);
	while(ros::ok())
	{
		steering_angle_msg.data = 21.37; //update komend skretu here
		steering_angle.publish(steering_angle_msg);
		target_speed_msg.data = 997.00;
		target_speed.publish(target_speed_msg);

		ros::spinOnce();
    rate.sleep();
	}
	return 0;
}
